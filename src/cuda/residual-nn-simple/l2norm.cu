#include "hip/hip_runtime.h"

#include "l2norm.h"

__global__ void l2NormKernel(const float *x, float *result, int n) {
    extern __shared__ float shared_data[]; // Shared memory for block-level reduction

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Each thread loads an element into shared memory
    shared_data[tid] = (i < n) ? x[i] * x[i] : 0.0f;
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    // First thread in each block writes the block's partial sum to result
    if (tid == 0) {
        atomicAdd(result, shared_data[0]);
    }
}

float l2Norm(const float *d_x, int n) {
    int threads_per_block = 256;
    int blocks = (n + threads_per_block - 1) / threads_per_block;

    float *d_result;
    float h_result = 0.0f;

    // Allocate device memory for result
    hipMalloc((void **)&d_result, sizeof(float));
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to calculate sum of squares
    l2NormKernel<<<blocks, threads_per_block, threads_per_block * sizeof(float)>>>(d_x, d_result, n);
    hipDeviceSynchronize();
    // Copy result back to host and compute square root
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    h_result = std::sqrt(h_result);

    // Clean up
    hipFree(d_result);

    return h_result;
}
